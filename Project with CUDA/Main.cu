#include "hip/hip_runtime.h"
#include "Runner.cuh"

void delay(int milliseconds) {
    clock_t start_time = clock();
    while (clock() < start_time + milliseconds);
}

__global__ void updatePositions(Runner* runners) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    runners[tid].updatePosition();
}

int main() {
    srand(time(NULL));
    Runner runners[NUM_RUNNERS];
    int numBlocks = 1;
    int threadsPerBlock = NUM_RUNNERS;
    int step = 1;
    int updateInterval = 1000 / TIME_INTERVAL; // Konum guncelleme araligi

    // Kosucularin konumunu ve hizini bellekte CUDA global hafizasina kopyalama
    Runner* d_runners;
    hipMalloc((void**)&d_runners, NUM_RUNNERS * sizeof(Runner));
    hipMemcpy(d_runners, runners, NUM_RUNNERS * sizeof(Runner), hipMemcpyHostToDevice);

    while (1) {
        // Kosucularin konumunu guncelleme
        dim3 numBlocks(1);
        dim3 threadsPerBlock(NUM_RUNNERS);
        updatePositions << <numBlocks, threadsPerBlock >> > (d_runners);
        hipDeviceSynchronize();

        // Kosucularin konumunu ve hizini CUDA global hafizasindan ana bellege kopyalama
        hipMemcpy(runners, d_runners, NUM_RUNNERS * sizeof(Runner), hipMemcpyDeviceToHost);

        // Bitis cizgisine ilk ulasan kosucunun indeksini bulma
        int winnerIndex = -1;
        for (int j = 0; j < NUM_RUNNERS; j++) {
            if (runners[j].position >= RACE_DISTANCE) {
                winnerIndex = j;
                break;
            }
        }

        // Yaris bittiginde her kosucunun guncel konumu
        if (winnerIndex != -1) {
            printf("Yaris bittiginde her kosucunun guncel konumu:\n");
            for (int j = 0; j < NUM_RUNNERS; j++) {
                printf("Kosucu %d: %.2f metre\n", j + 1, runners[j].position);
            }
            break;
        }

        if (step % updateInterval == 0) {
            printf("Kosucularin anlik konumu (Saniye: %d):\n", step);
            for (int j = 0; j < NUM_RUNNERS; j++) {
                printf("Kosucu %d: %.2f metre\n", j + 1, runners[j].position);
            }
            printf("\n");
        }

        delay(TIME_INTERVAL);
        step++;
    }

    // Yarisin siralamasini hesaplama
    int sortedIndices[NUM_RUNNERS];
    for (int i = 0; i < NUM_RUNNERS; i++) {
        sortedIndices[i] = i;
    }

    // Siralama islemi
    for (int i = 0; i < NUM_RUNNERS - 1; i++) {
        for (int j = i + 1; j < NUM_RUNNERS; j++) {
            if (runners[sortedIndices[i]].position < runners[sortedIndices[j]].position) {
                int temp = sortedIndices[i];
                sortedIndices[i] = sortedIndices[j];
                sortedIndices[j] = temp;
            }
        }
    }

    // Yarisin siralamasini yazdirma
    printf("Yarisin siralamasi:\n");
    for (int i = 0; i < NUM_RUNNERS; i++) {
        printf("Sira %d: Kosucu %d\n", i + 1, sortedIndices[i] + 1);
    }

    // Bellek temizleme
    hipFree(d_runners);

    return 0;
}